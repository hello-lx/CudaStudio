
#include <hip/hip_runtime.h>
#include<iostream>
#include<math.h>
#include<sys/time.h>

// 普通加法gpu和cpu对比加速

// 编译： nvcc gpu_code.cu -o gpu_code
// 运行： ./gpu_code


__global__
void add(int n, float *x, float *y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i=index; i<n; i+=stride)
        y[i] = x[i] + y[i];
}

int main(int argc, char const *argv[])
{
    int N = 1 << 20;
    float *x;
    float *y;

    // Allocate Unified Memory – accessible from CPU or GPU
    // 内存分配，在GPU或者CPU上统一分配内存
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    for(int i=0; i<N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    struct timeval t1, t2;
    double timeuse;
    gettimeofday(&t1, NULL);


    dim3 blockSize(256);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
    add<<<gridSize, blockSize>>>(N, x, y);  // 单线程运行

    // 实例化CUDA event
	hipEvent_t e_start, e_stop;
	//创建事件
	hipEventCreate(&e_start);
	hipEventCreate(&e_stop);
	
	//记录事件，开始计算时间
	hipEventRecord(e_start, 0);

    // Wait for GPU to finish before accessing on host
    // CPU需要等待cuda上的代码运行完毕，才能对数据进行读取
    // cudaDeviceSynchronize();

    //记录结束时事件
	hipEventRecord(e_stop, 0);// 0 代表CUDA流0
	//等待事件同步后
	hipEventSynchronize(e_stop);
	//计算对应的时间，评估代码性能
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, e_start, e_stop);

    std::cout << "add(int, float*, float*) gpu time: " << elapsedTime << "ms" << std::endl;


    // gettimeofday(&t2, NULL);
    // timeuse = (t2.tv_sec, t1.tv_sec) + (double)(t2.tv_usec - t1.tv_usec) / 1000.0;
    // std::cout << "add(int, float*, float*) time: " << timeuse << "ms" << std::endl;

    float maxError = 0.0f;
    for(int i=0; i<N; i++)
        maxError = fmax(maxError, fabs(3.0f - y[i]));
    std::cout << "Max error: " << maxError << std::endl;
    
    // free memory
    hipFree(x);
    hipFree(y);

    return 0;
}

